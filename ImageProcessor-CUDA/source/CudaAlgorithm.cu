#include "hip/hip_runtime.h"
#include "CudaAlgorithm.hpp"
#include "HelperFunctions.hpp"
#include <cstdlib>
#include <SFML/Graphics.hpp>
#include <hip/hip_runtime.h>
#include <>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

__device__ void alignChannel(int& channelValue)
{
	channelValue = (channelValue > 255) ? 255 : channelValue;
	channelValue = (channelValue < 0) ? 0 : channelValue;
}

__global__ void applyFilterOnCuda(
	const sf::Uint8* inputImageData, sf::Uint8* outputImageData,
	const std::size_t width, const std::size_t height,
	const float* filter, const int kernelSize)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int kernelMargin = kernelSize / 2;


	if (((x - kernelMargin) > 0 && (x + kernelMargin) < width) &&
		((y - kernelMargin) > 0 && (y + kernelMargin) < height))
	{
		int newRedChannel{}, newGreenChannel{}, newBlueChannel{};
		for (int kernelX = -kernelMargin; kernelX <= kernelMargin; ++kernelX)
		{
			for (int kernelY = -kernelMargin; kernelY <= kernelMargin; ++kernelY)
			{
				const auto kernelXIndex = kernelX + kernelMargin;
				const auto kernelYIndex = kernelY + kernelMargin;
				const auto kernelIndex = kernelXIndex * kernelSize + kernelYIndex;
				const auto kernelValue = filter[kernelIndex];
				
				const auto inPixel = &inputImageData[((x + kernelX) + (y + kernelY) * width) * 4];
				newRedChannel += inPixel[0] * kernelValue;
				newGreenChannel += inPixel[1] * kernelValue;
				newBlueChannel += inPixel[2] * kernelValue;
			}
		}

		alignChannel(newRedChannel);
		alignChannel(newGreenChannel);
		alignChannel(newBlueChannel);

		auto outPixel = &outputImageData[(x + y * width) * 4];
		outPixel[0] = newRedChannel;
		outPixel[1] = newGreenChannel;
		outPixel[2] = newBlueChannel;
	}
}

void Cuda::applyFilter(sf::Image& image, const Filter::Kernel& filter)
{
	thrust::host_vector<sf::Uint8> hostImageData{ image.getPixelsPtr(), image.getPixelsPtr() + calculateImageSize(image) };
	thrust::device_vector<sf::Uint8> devImageData(calculateImageSize(image));
	thrust::device_vector<sf::Uint8> devOutputImageData(calculateImageSize(image));
	thrust::copy(hostImageData.begin(), hostImageData.end(), devImageData.begin());

	thrust::device_vector<float> devKernel{};
	for (const auto& filterRow : filter)
	{
		devKernel.insert(devKernel.end(), filterRow.begin(), filterRow.end());
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(ceil((float)image.getSize().x / threadsPerBlock.x), ceil((float)image.getSize().y / threadsPerBlock.y));
	applyFilterOnCuda<<<numBlocks, threadsPerBlock>>>(
		devImageData.data().get(), devOutputImageData.data().get(),
		image.getSize().x, image.getSize().y,
		devKernel.data().get(), filter.size());

	hipEventRecord(stop);
	hipDeviceSynchronize();
	hipEventSynchronize(stop);

	float timeMs{};
	hipEventElapsedTime(&timeMs, start, stop);
	std::cout << "[CUDA] (only CUDA calculations): " << timeMs << " ms" << std::endl;

	thrust::copy(devOutputImageData.begin(), devOutputImageData.end(), hostImageData.begin());
	image.create(image.getSize().x, image.getSize().y, hostImageData.data());
}